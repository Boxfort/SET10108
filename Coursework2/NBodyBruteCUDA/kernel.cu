#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

const unsigned int N	   = 100;				// Number of bodies.
const unsigned int ITERS   = 1000;				// Number of simulation iterations.
const unsigned int THREADS = 32;				// Number of threads per block.
const unsigned int BLOCKS  = ceil(N / THREADS);	// Number of blocks required to satisfy N bodies with THREADS threads per block.

void cuda_info()
{
	// Get cuda device;
	int device;
	hipGetDevice(&device);

	// Get device properties
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	//Display Properties
	cout << "Name: " << properties.name << endl;
	cout << "CUDA Capability: " << properties.major << endl;
	cout << "Cores: " << properties.multiProcessorCount << endl;
	cout << "Memory: " << properties.totalGlobalMem / (1024 * 1024) << "MB" << endl;
	cout << "Clock freq: " << properties.clockRate / 1000 << "MHz" << endl;
}

__global__ void n_body(unsigned int *iterations, float *pi, hiprandState *state)
{
	// Calculate index
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	unsigned int start = idx * iterations[0];
	unsigned int end = start + iterations[0];
}

void initBodies(float2* pos, float2* vel, float* mass)
{
	float* random_pos;
	float* random_mass;
	hipHostMalloc((void **)&random_pos,  (sizeof(float) * N) * 2);
	hipHostMalloc((void **)&random_mass, (sizeof(float) * N));

	hiprandGenerator_t rnd;
	hiprandCreateGenerator(&rnd, HIPRAND_RNG_QUASI_SOBOL32);
	hiprandSetQuasiRandomGeneratorDimensions(rnd, 1);
	hiprandSetGeneratorOrdering(rnd, HIPRAND_ORDERING_QUASI_DEFAULT);

	hiprandGenerateUniform(rnd, (float*)random_pos, N * 2);
	hiprandGenerateUniform(rnd, (float*)random_mass, N);

	for (int i = 1; i < N; i++)
	{
		pos[i].x = (2.0f * random_pos[i]) - 1.0f; // Init at position between -1 : 1
		pos[i].y = (2.0f * random_pos[i + 1]) - 1.0f; // Init at position between -1 : 1
		vel[i].x = 0;
		vel[i].y = 0;
		mass[i]  = static_cast<int>(random_mass[i]) % (500 - 100 + 1) + 100;
	}

	pos[0].x = 0;
	pos[0].y = 0;
	vel[0].x = 0.0;
	vel[0].y = 0.0;
	mass[0]  = 3000;
}

int main()
{
	//Init CUDA - select device
	hipSetDevice(0);
	cuda_info();

	// Declare host memory 
	float2 *host_pos;			// out
	float2 *host_vel;			// out
	float  *host_mass;			// in
	int	   *host_iterations;	// in

	// Allocate host memory
	hipHostMalloc((void **)&host_pos,		  (sizeof(float2) * N) * ITERS);
	hipHostMalloc((void **)&host_vel,		  (sizeof(float2) * N) * ITERS);
	hipHostMalloc((void **)&host_mass,		  (sizeof(float) * N) * ITERS);
	hipHostMalloc((void **)&host_iterations, (sizeof(unsigned int)));

	// Initialise host memory
	initBodies(host_pos, host_vel, host_mass);
	host_iterations[0] = ITERS;

	// Declare device memory
	float2		 *dev_pos;			// out
	float2		 *dev_vel;			// out
	float		 *dev_mass;			// in
	unsigned int *dev_iterations;	// in

	// Allocate device memory
	hipMalloc((void**)&dev_pos,		(sizeof(float2) * N) * ITERS);
	hipMalloc((void**)&dev_vel,		(sizeof(float2) * N) * ITERS);
	hipMalloc((void**)&dev_mass,		(sizeof(float) * N)  * ITERS);
	hipMalloc((void**)&dev_iterations, (sizeof(unsigned int)));

	//Copy memory from host to device
	hipMemcpy(dev_pos,		   &host_pos[0],		(sizeof(float2) * N) * ITERS, hipMemcpyHostToDevice);
	hipMemcpy(dev_vel,		   &host_vel[0],		(sizeof(float2) * N) * ITERS, hipMemcpyHostToDevice);
	hipMemcpy(dev_mass,	   &host_mass[0],		(sizeof(float) * N)  * ITERS, hipMemcpyHostToDevice);
	hipMemcpy(dev_iterations, &host_iterations[0], (sizeof(unsigned int)),		  hipMemcpyHostToDevice);

	// Execute Kernel
	n_body << <BLOCKS, THREADS >> >(dev_pos, dev_vel, dev_mass, dev_iterations);

	// Wait for kernal to complete
	hipDeviceSynchronize();

	// Read output buffer to host
	hipMemcpy(&host_pos[0], dev_pos, (sizeof(float2) * N) * ITERS, hipMemcpyDeviceToHost);
	hipMemcpy(&host_vel[0], dev_vel, (sizeof(float2) * N) * ITERS, hipMemcpyDeviceToHost);

	hipFree(dev_pos);
	hipFree(dev_vel);
	hipFree(dev_mass);
	hipFree(dev_iterations);

	int a;
	cin >> a;

	return 0;
}
