#include "hip/hip_runtime.h"
#include "stdafx.h"
#include <hip/hip_runtime.h>
#include <>
#include <iostream>
#include <vector>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

using namespace std;

void cuda_info()
{
	// Get cuda device;
	int device;
	hipGetDevice(&device);

	// Get device properties
	hipDeviceProp_t properties;
	hipGetDeviceProperties(&properties, device);

	//Display Properties
	cout << "Name: " << properties.name << endl;
	cout << "CUDA Capability: " << properties.major << endl;
	cout << "Cores: " << properties.multiProcessorCount << endl;
	cout << "Memory: " << properties.totalGlobalMem / (1024 * 1024) << "MB" << endl;
	cout << "Clock freq: " << properties.clockRate / 1000 << "MHz" << endl;
}

__global__ void monte_carlo_pi(unsigned int *iterations, float *pi, hiprandState *state)
{
	// Calculate index
	unsigned int idx = (blockIdx.x * blockDim.x) + threadIdx.x;

	unsigned int start = idx * iterations[0];
	unsigned int end = start + iterations[0];

	hiprand_init(111, idx, 0, &state[idx]);  // Initialize CURAND

	//Set start result to 0
	unsigned int points_in_circle = 0;

	for (unsigned int i = start; i < end; ++i)
	{
		//Get point to work on
		float2 point = make_float2(hiprand_uniform(&state[idx]), hiprand_uniform(&state[idx]));
		// Calculate length
		float l = sqrtf((point.x * point.x) + (point.y * point.y));
		// Check if length and add to result accordingly
		if (l <= 1.0f)
			++points_in_circle;
	}

	pi[idx] = 4.0f * points_in_circle / (float)iterations[0]; // return estimate of pi
}

int main()
{
	const unsigned int POINTS = pow(2, 24);
	const unsigned int THREADS = 256;
	const unsigned int POINTS_PER_THREAD = POINTS / THREADS;

	//Init CUDA - select device
	hipSetDevice(0);
	cuda_info();

	// Create host memory
	auto data_size = sizeof(float) * THREADS;
	hiprandState *devStates;
	vector<float> pi_values(THREADS); // Out

	// Declare buffers
	float *buffer_pi_values;
	unsigned int *buffer_points_per_thread;

	// Init Buffers
	hipMalloc((void**)&buffer_pi_values, data_size);
	hipMalloc((void**)&buffer_points_per_thread, sizeof(unsigned int));
	hipMalloc((void**)&devStates, THREADS * sizeof(hiprandState));

	//Copy memory from host to device
	hipMemcpy(buffer_pi_values, &pi_values[0], data_size, hipMemcpyHostToDevice);
	hipMemcpy(buffer_points_per_thread, &POINTS_PER_THREAD, sizeof(unsigned int), hipMemcpyHostToDevice);

	// Execute Kernel
	monte_carlo_pi << <1, THREADS >> >(buffer_points_per_thread, buffer_pi_values, devStates);

	// Wait for kernal to complete
	hipDeviceSynchronize();

	// Read output buffer to host
	hipMemcpy(&pi_values[0], buffer_pi_values, data_size, hipMemcpyDeviceToHost);

	float pi_estimate = 0.0f;

	for (int i = 0; i < THREADS; i++)
	{
		pi_estimate += pi_values[i];
	}

	 pi_estimate /= THREADS;

	cout << "pi = " << pi_estimate << endl;

	hipFree(buffer_pi_values);
	hipFree(buffer_points_per_thread);
	hipFree(devStates);

	int a;
	cin >> a;

	return 0;
}
